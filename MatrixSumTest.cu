/**
 * Sum two square matrix (A = B + C)
 * Exercise 3.1 of programming massively parallel processors book
 * Solution provided with matrix view as array
 * @author Niccolò Bellaccini
 */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h> //rand
#include <time.h> //rand
#include <math.h> //ceil

#include <iostream> //std::cerr

#define BLOCK_WIDTH 64

void initData(float* M, int nRows){ //Remeber: matrix square

	for (int i=0; i<nRows; i++){
		for(int j=0; j<nRows; j++){
			M[nRows * i + j] = (float) (rand() & 0xFF) / 10.0f;
		}
	}
}

void displayData(float *M, int nRows){

	for (int i=0; i<nRows; i++){
		printf("\n");
		for(int j=0; j<nRows; j++){
			printf("%.1f\t", M[nRows * i + j]);
		}
	}
}

/**
 * function-like macro
 * __LINE__ = contains the line number of the currently compiled line of code
 * __FILE__ = string that contains the name of the source file being compiled
 * # operator = turns the argument it precedes into a quoted string
 * Reference: [C the complete reference]
 * check with > nvcc -E
 */
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__, __LINE__, #value, value)

static void CheckCudaErrorAux (const char *file, unsigned line,
		const char *statement, hipError_t err){
	if(err == hipSuccess)
		return;
	std::cerr << statement << " returned " << hipGetErrorString(err) <<
			"(" << err << ") at " << file << ":" << line << std::endl;
	exit(1);
}


//Kernel function (point b of exercise)
__global__ void matrixAddKernel(float *A, float *B, float *C, int nRows){
	int size = nRows * nRows; //Remember: square matrices
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<size)
		A[i] = B[i] + C[i];
}

//Kernel function (point c of exercise)
__global__ void matrixPerRowsAddKernel(float *A, float *B, float *C, int nRows){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<nRows){
		for (int j=0; j<nRows; j++){
			A[i * nRows + j] = B[i * nRows + j] + C[i * nRows + j];
		}
	}
}

//Kernel function (point d of exercise)
__global__ void matrixPerColumnsAddKernel(float *A, float *B, float *C, int nColumns){
	int i = threadIdx.x + blockDim.x * blockIdx.x;
	if (i<nColumns){
		for (int j=0; j<nColumns; j++){
			A[i + j * nColumns] = B[i + j * nColumns] + C[i + j * nColumns];
		}
	}
}

/**
 * Stub function used to compute matrices sum.
 * (function used to launch the kernel and to allocate device mem, ...)
 */
void matrixAdd(float* A, float *B, float *C, int nRows){

	size_t size = nRows * nRows * sizeof(float);
	float * d_A;
	float * d_B;
	float * d_C;

	//Allocate device memory for matrices
	CUDA_CHECK_RETURN(hipMalloc((void **) &d_B, size));
	CUDA_CHECK_RETURN(hipMalloc((void **) &d_C, size));
	CUDA_CHECK_RETURN(hipMalloc((void **) &d_A, size));

	//Copy B and C to device memory
	CUDA_CHECK_RETURN(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));
	CUDA_CHECK_RETURN(hipMemcpy(d_C, C, size, hipMemcpyHostToDevice));

	//Kernel launch code
	//assume block of 64 threads
	matrixAddKernel <<< ceil((double)(nRows*nRows)/BLOCK_WIDTH), BLOCK_WIDTH>>>(d_A, d_B, d_C, nRows);

	//Two other possible kernel functions

	//matrixPerRowsAddKernel<<< ceil((double)nRows/BLOCK_WIDTH) ,BLOCK_WIDTH>>>(d_A, d_B, d_C, nRows);
	//matrixPerColumnsAddKernel<<< ceil((double)nRows/BLOCK_WIDTH) ,BLOCK_WIDTH>>>(d_A, d_B, d_C, nRows);

	//Copy A from the device memory
	CUDA_CHECK_RETURN(hipMemcpy(A, d_A, size, hipMemcpyDeviceToHost));

	//Free device matrices
	hipFree(d_C);
	hipFree(d_B);
	hipFree(d_A);
}

int main(int argc, char** argv){

	//Initialize random seed
	//@see http://www.cplusplus.com/reference/cstdlib/srand/
	//@see https://stackoverflow.com/questions/20158841/my-random-number-generator-function-generates-the-same-number
    srand(time(NULL));

	int numRows;
	printf("\nInsert the number of rows (equivalently columns): ");
	scanf("%d", &numRows);

	int numColumns = numRows; //Square matrix
	int nElem = numRows * numColumns;
	float * B = (float *) malloc(nElem * sizeof(float));
	float * C = (float *) malloc(nElem * sizeof(float));

	float * A = (float *) malloc(nElem * sizeof(float));

	//Initialize B and C matrices
	initData(B, numRows);
	initData(C, numRows);

	//Display B and C matrices
	printf("\n\tMatrice B\n");
	displayData(B, numRows);
	printf("\n\n\tMatrice C\n");
	displayData(C, numRows);

	//matrices sum
	matrixAdd(A, B, C, numRows);

	//Display A matrix
	printf("\n\n\tMatrice A\n");
	displayData(A, numRows);
}
